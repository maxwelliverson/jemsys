#include "hip/hip_runtime.h"
//
// Created by maxwe on 2021-10-30.
//

#include "rng.hpp"

#include <cuda/barrier>
#include <cuda/atomic>
#include <cuda/std/chrono>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>


#include <hiprand.h>



namespace cg = cooperative_groups;

extern __shared__ unsigned           tmpResults[];
extern __shared__ unsigned           scratchU32[];
extern __shared__ float              scratchFloats[];
extern __shared__ double             scratchDoubles[];
extern __shared__ unsigned long long scratchU64[];

namespace {

  template <typename State>
  __global__ void generate_32bit_kernel(State* globalState,
                                        size_t n,
                                        size_t sharedMemorySize,
                                        hipDeviceptr_t out) noexcept {

    __shared__ cuda::barrier<cuda::thread_scope_block> barrier[1];

    size_t id     = cg::thread_block::thread_rank();
    unsigned scale = sharedMemorySize / cg::thread_block::size();
    unsigned iterations = n / sharedMemorySize;
    size_t offset = id * scale;
    auto outPtr = reinterpret_cast<char*>(out) + offset;

    if (id == 0)
      init(barrier, cg::thread_block::size());

    State localState = globalState[id];

    for (unsigned i = 0; i < iterations; ++i) {

      for (unsigned j = 0; j < scale; ++j)
        scratchU32[offset + j] = hiprand(&localState);

      cuda::memcpy_async(cg::this_thread_block(), outPtr + i * sharedMemorySize, scratchU32, scale, barrier[0]);

      barrier->arrive_and_wait();

      // cuda::memcpy_async(block, out, tmpResults, n, barrier);

    }

    globalState[id] = localState;
  }

  template <typename State>
  __global__ void generate_normal_kernel(State* globalState,
                                         size_t n,
                                         size_t sharedMemorySize,
                                         hipDeviceptr_t out) noexcept {

    __shared__ cuda::barrier<cuda::thread_scope_block> barrier[1];

    size_t id     = cg::thread_block::thread_rank();
    unsigned scale = sharedMemorySize / cg::thread_block::size();
    unsigned iterations = n / sharedMemorySize;
    size_t offset = id * scale;
    auto outPtr = reinterpret_cast<char*>(out) + offset;

    if (id == 0)
      init(barrier, cg::thread_block::size());

    State localState = globalState[id];

    for (unsigned i = 0; i < iterations; ++i) {

      for (unsigned j = 0; j < scale; ++j)
        scratchFloats[offset + j] = hiprand_normal(&localState);

      cuda::memcpy_async(cg::this_thread_block(), outPtr + i * sharedMemorySize, tmpResults, scale, barrier[0]);

      barrier->arrive_and_wait();

      // cuda::memcpy_async(block, out, tmpResults, n, barrier);

    }

    globalState[id] = localState;
  }

  template <typename State>
  __global__ void generate_uniform_kernel(State* globalState,
                                          size_t n,
                                          size_t sharedMemorySize,
                                          hipDeviceptr_t out) noexcept {

    __shared__ cuda::barrier<cuda::thread_scope_block> barrier[1];

    size_t id     = cg::thread_block::thread_rank();
    unsigned scale = sharedMemorySize / cg::thread_block::size();
    unsigned iterations = n / sharedMemorySize;
    size_t offset = id * scale;
    auto outPtr = reinterpret_cast<char*>(out) + offset;

    if (id == 0)
      init(barrier, cg::thread_block::size());

    State localState = globalState[id];

    for (unsigned i = 0; i < iterations; ++i) {

      for (unsigned j = 0; j < scale; ++j)
        scratchFloats[offset + j] = hiprand_uniform(&localState);

      cuda::memcpy_async(cg::this_thread_block(), outPtr + i * sharedMemorySize, tmpResults, scale, barrier[0]);

      barrier->arrive_and_wait();

      // cuda::memcpy_async(block, out, tmpResults, n, barrier);

    }

    globalState[id] = localState;
  }

  template <typename State>
  __global__ void generate_log_normal_kernel(State* globalState,
                                              float mean,
                                              float stddev,
                                              size_t n,
                                              size_t sharedMemorySize,
                                              hipDeviceptr_t out) noexcept {

    __shared__ cuda::barrier<cuda::thread_scope_block> barrier[1];

    size_t id     = cg::thread_block::thread_rank();
    unsigned scale = sharedMemorySize / cg::thread_block::size();
    unsigned iterations = n / sharedMemorySize;
    size_t offset = id * scale;
    auto outPtr = reinterpret_cast<char*>(out) + offset;

    if (id == 0)
      init(barrier, cg::thread_block::size());

    State localState = globalState[id];

    for (unsigned i = 0; i < iterations; ++i) {

      for (unsigned j = 0; j < scale; ++j)
        scratchFloats[offset + j] = hiprand_log_normal(&localState, mean, stddev);

      cuda::memcpy_async(cg::this_thread_block(), outPtr + i * sharedMemorySize, tmpResults, scale, barrier[0]);

      barrier->arrive_and_wait();

      // cuda::memcpy_async(block, out, tmpResults, n, barrier);

    }

    globalState[id] = localState;
  }

  void hostCallbackFn(void* arg) noexcept {
    auto msg = static_cast<agt_message_t*>(arg);
    auto args = static_cast<rng::refill_block_args*>(msg->payload);
    args->block->reset();
    agt_slot_t slot;
    agt_mailbox_acquire_slot(args->responseMailbox, &slot, sizeof(rng::return_block_args), JEM_WAIT);
    slot.id = 0;
    new(slot.payload) rng::return_block_args{
      .block = args->block
    };
    agt_send(&slot, args->serverSelf, AGT_IGNORE_RESULT);
    agt_return(msg, AGT_SUCCESS);
  }

  class xorwow_generator : public rng::generator {
    hiprandStateXORWOW_t* globalState;
  public:
    void generate_32bit(const rng::generate_args& args) noexcept override {
      generate_32bit_kernel<<<args.blockSize, args.gridSize, args.sharedMemSize, args.stream>>>(globalState, args.bufferSize, args.sharedMemSize, args.deviceBuffer);
    }
    void generate_discrete(const rng::generate_args& args, hiprandDiscreteDistribution_t discreteDistribution) noexcept override {

    }

    void generate_poisson(const rng::generate_args& args, double lambda) noexcept override {

    }


    void generate_normal(const rng::generate_args& args) noexcept override {
      generate_normal_kernel<<<args.blockSize, args.gridSize, args.sharedMemSize, args.stream>>>(globalState, args.bufferSize, args.sharedMemSize, args.deviceBuffer);
    }
    void generate_uniform(const rng::generate_args& args) noexcept override {
      generate_uniform_kernel<<<args.blockSize, args.gridSize, args.sharedMemSize, args.stream>>>(globalState, args.bufferSize, args.sharedMemSize, args.deviceBuffer);
    }
    void generate_log_normal(const rng::generate_args& args, float mean, float stddev) noexcept override {
      generate_log_normal_kernel<<<args.blockSize, args.gridSize, args.sharedMemSize, args.stream>>>(globalState, mean, stddev, args.bufferSize, args.sharedMemSize, args.deviceBuffer);
    }


    void generate_normal_double(const rng::generate_args& args) noexcept override {}
    void generate_uniform_double(const rng::generate_args& args) noexcept override {}
    void generate_log_normal_double(const rng::generate_args& args, double mean, double stddev) noexcept override {}
  };



  class xorwow_engine : public rng::random_engine {

    hiprandStateXORWOW_t* globalState;


  public:

    void refill(const rng::generate_args& args) noexcept override {
      rng::kernels::generate<<<args.blockSize, args.gridSize, args.sharedMemSize, args.stream>>>(globalState, args.bufferSize, args.sharedMemSize, args.deviceBuffer);
      /*auto memcpyResult = hipMemcpyDtoHAsync(blk->base, deviceBuffer, bufferSize, stream);
      auto callbackResult = hipLaunchHostFunc(stream, hostCallbackFn, msg);*/
    }

    void anchor() noexcept override {}
  };

  class sobol32_engine : public rng::random_engine {

    hiprandStateSobol32_t* globalState;


  public:

    void refill(const rng::generate_args& args) noexcept override {
      rng::kernels::generate<<<args.blockSize, args.gridSize, args.sharedMemSize, args.stream>>>(globalState, args.bufferSize, args.sharedMemSize, args.deviceBuffer);
      /*auto memcpyResult = hipMemcpyDtoHAsync(blk->base, deviceBuffer, bufferSize, stream);
      auto callbackResult = hipLaunchHostFunc(stream, hostCallbackFn, msg);*/
    }

    void anchor() noexcept override {}
  };

  class sobol64_engine : public rng::random_engine {

    hiprandStateSobol64_t* globalState;


  public:

    void refill(const rng::generate_args& args) noexcept override {
      rng::kernels::generate<<<args.blockSize, args.gridSize, args.sharedMemSize, args.stream>>>(globalState, args.bufferSize, args.sharedMemSize, args.deviceBuffer);
      /*auto memcpyResult = hipMemcpyDtoHAsync(blk->base, deviceBuffer, bufferSize, stream);
      auto callbackResult = hipLaunchHostFunc(stream, hostCallbackFn, msg);*/
    }

    void anchor() noexcept override {}
  };

  class scrambled_sobol32_engine : public rng::random_engine {

    hiprandStateScrambledSobol32_t* globalState;


  public:

    void refill(const rng::generate_args& args) noexcept override {
      rng::kernels::generate<<<args.blockSize, args.gridSize, args.sharedMemSize, args.stream>>>(globalState, args.bufferSize, args.sharedMemSize, args.deviceBuffer);
      /*auto memcpyResult = hipMemcpyDtoHAsync(blk->base, deviceBuffer, bufferSize, stream);
      auto callbackResult = hipLaunchHostFunc(stream, hostCallbackFn, msg);*/
    }

    void anchor() noexcept override {}
  };

  class scrambled_sobol64_engine : public rng::random_engine {

    hiprandStateScrambledSobol64_t* globalState;


  public:

    void refill(const rng::generate_args& args) noexcept override {
      rng::kernels::generate<<<args.blockSize, args.gridSize, args.sharedMemSize, args.stream>>>(globalState, args.bufferSize, args.sharedMemSize, args.deviceBuffer);
      /*auto memcpyResult = hipMemcpyDtoHAsync(blk->base, deviceBuffer, bufferSize, stream);
      auto callbackResult = hipLaunchHostFunc(stream, hostCallbackFn, msg);*/
    }

    void anchor() noexcept override {}
  };
}

void rng::generator::anchor() noexcept {}




template <typename State>
__global__ void rng::kernels::generate(State* globalState,
                                       size_t n,
                                       size_t sharedMemorySize,
                                       hipDeviceptr_t out) noexcept {

  __shared__ cuda::barrier<cuda::thread_scope_block> barrier[1];

  size_t id     = cg::thread_block::thread_rank();
  unsigned scale = sharedMemorySize / cg::thread_block::size();
  unsigned iterations = n / sharedMemorySize;
  size_t offset = id * scale;
  auto outPtr = reinterpret_cast<char*>(out) + offset;

  if (id == 0)
    init(barrier, cg::thread_block::size());

  State localState = globalState[id];

  for (unsigned i = 0; i < iterations; ++i) {

    for (unsigned j = 0; j < scale; ++j)
      tmpResults[offset + j] = hiprand(&localState);

    cuda::memcpy_async(cg::this_thread_block(), outPtr + i * sharedMemorySize, tmpResults, scale, barrier[0]);

    barrier->arrive_and_wait();

    // cuda::memcpy_async(block, out, tmpResults, n, barrier);

  }

  globalState[id] = localState;
}


template __global__ void rng::kernels::generate(hiprandStateXORWOW_t* globalState,
                                                size_t n,
                                                size_t sharedMemorySize,
                                                hipDeviceptr_t out) noexcept;
template __global__ void rng::kernels::generate(hiprandStateSobol32_t* globalState,
                                                size_t n,
                                                size_t sharedMemorySize,
                                                hipDeviceptr_t out) noexcept;
template __global__ void rng::kernels::generate(hiprandStateSobol64_t* globalState,
                                                size_t n,
                                                size_t sharedMemorySize,
                                                hipDeviceptr_t out) noexcept;
template __global__ void rng::kernels::generate(hiprandStateScrambledSobol32_t* globalState,
                                                size_t n,
                                                size_t sharedMemorySize,
                                                hipDeviceptr_t out) noexcept;
template __global__ void rng::kernels::generate(hiprandStateScrambledSobol64_t* globalState,
                                                size_t n,
                                                size_t sharedMemorySize,
                                                hipDeviceptr_t out) noexcept;
template __global__ void rng::kernels::generate(hiprandStateMtgp32_t* globalState,
                                                size_t n,
                                                size_t sharedMemorySize,
                                                hipDeviceptr_t out) noexcept;
template __global__ void rng::kernels::generate(hiprandStateMRG32k3a_t* globalState,
                                                size_t n,
                                                size_t sharedMemorySize,
                                                hipDeviceptr_t out) noexcept;
template __global__ void rng::kernels::generate(hiprandStatePhilox4_32_10_t* globalState,
                                                size_t n,
                                                size_t sharedMemorySize,
                                                hipDeviceptr_t out) noexcept;





